
#include <hip/hip_runtime.h>
#define INFINITY __int_as_float(0x7f800000)

extern "C" {
    __device__ void get_sample_params(const float **discrete_spike_times,
                                      const float **spike_times,
                                      const float **exp_tau_s,
                                      const float **exp_tau,
                                      const float **spike_weights,
                                      int n_neurons, int sample_idx, int neuron_idx, int max_n_pre_spike) {
        int sample_start_idx = sample_idx * max_n_pre_spike;

        *discrete_spike_times += sample_start_idx;
        *spike_times += sample_start_idx;
        *exp_tau_s += sample_start_idx;
        *exp_tau += sample_start_idx;
        *spike_weights += (sample_idx * n_neurons + neuron_idx) * max_n_pre_spike;
    }

    __device__ void get_neuron_results(int **n_spikes,
                                       float **a,
                                       float **x,
                                       float **spike_times,
                                       float **discrete_spike_times,
                                       float **post_exp_tau,
                                       int n_neurons, int sample_idx, int neuron_idx, int max_n_post_spike) {
        int sample_neuron_idx = (sample_idx * n_neurons + neuron_idx);
        int res_start_idx = sample_neuron_idx * max_n_post_spike;

        *n_spikes += sample_neuron_idx;
        *a += res_start_idx; // Assuming the spike numbers is the same
        *x += res_start_idx; // Might have a problem as we miss more discrete spikes
        *spike_times += res_start_idx;
        *discrete_spike_times += res_start_idx;
        *post_exp_tau += res_start_idx;
    }

    __device__ bool compute_spikes(const float c,
                                   int *n_spikes,
                                   float *a,
                                   float *x,
                                   float *spike_times,
                                   float *discrete_spike_times,
                                   float *post_exp_tau,
                                   float cumul_a,
                                   float *cumul_b,
                                   float last_spike,
                                   float discrete_last_spike,
                                   float next_spike,
                                   float delta_theta_tau,
                                   float tau,
                                   float time_delta,
                                   float max_simulation,
                                   int neuron_idx,
                                   int max_n_post_spike,
                                   int sample_idx) {
        float x_tmp, inside_log, tmp, tmp_disc;

        // Compute until there is no spike anymore
        while (true) {
            tmp = (*cumul_b) * (*cumul_b) - 4.0 * cumul_a * c;

            if (tmp < 0) // Negative discriminant, no spike --> stop
                return false;
            x_tmp = sqrtf(tmp);
            tmp = x_tmp + (*cumul_b); // X_k + b_k

            if (tmp == 0.0) // Division per zero, no spike --> stop
                return false;
            inside_log = 2 * cumul_a / tmp;
            if (inside_log < 0) // Negative log, no spike --> stop
                return false;

            tmp = tau * __logf(inside_log); // t_k
            
            float potential = tau * ((*cumul_b) * __expf(-tmp / tau) - cumul_a * __expf(-2 * tmp / tau));
            
            // 0.0001 can be adjusted depending on desired margin for error
            if (potential + 0.0001 < (c * tau)) {
            	return false;
            }

            // Add timeshift
//             tmp = tmp + ((3 * time_delta) / 4) ;
            tmp_disc = tmp;

            // Discretize to next time step
            if (time_delta != 0.0f)
                tmp_disc = tmp_disc + (time_delta - fmodf(tmp, time_delta));
            else
                tmp_disc = tmp;

            // Spike time is before the last pre-spike or after the next spike --> stop
            if (tmp_disc <= discrete_last_spike || tmp_disc > max_simulation || tmp_disc > next_spike)
                return false;

            discrete_spike_times[*n_spikes] = tmp_disc;
            // Spike time is valid
            //printf("%f %f\n", tmp, discrete_spike_times[*n_spikes]);
            a[*n_spikes] = cumul_a;
            x[*n_spikes] = x_tmp;
            spike_times[*n_spikes] = tmp;
            discrete_last_spike = discrete_spike_times[*n_spikes]; // we use the discrete spikes
            last_spike = tmp;
            post_exp_tau[*n_spikes] = inside_log;
            *cumul_b -= delta_theta_tau * inside_log; // Apply reset to b
            (*n_spikes)++;
            if (*n_spikes >= max_n_post_spike) {
                return true;
            }
        }
    }

    __global__ void compute_spike_times_kernel(// Parameters
                                               const float *discrete_spike_times,
                                               const float *spike_times,
                                               const float *exp_tau_s,
                                               const float *exp_tau,
                                               const float *spike_weights,
                                               const float c,
                                               float delta_theta_tau,
                                               float tau,
                                               float time_delta,
                                               float max_simulation,
                                               int max_n_pre_spike,
                                               int max_n_post_spike,
                                               // Outputs	
                                               int *n_spikes,
                                               float *a,
                                               float *x,
                                               float *out_spike_times,
                                               float *out_discrete_spike_times,
                                               float *post_exp_tau) {
        int n_neurons = gridDim.x;
        int sample_idx = threadIdx.x;
        int neuron_idx = blockIdx.x;

        //if(spike_times[0] != 0 )
            //printf("%f %f\n", spike_times[0], discrete_spike_times[0]);

        get_sample_params(&discrete_spike_times, &spike_times, &exp_tau_s, &exp_tau, &spike_weights,
                          n_neurons, sample_idx, neuron_idx, max_n_pre_spike);
        get_neuron_results(&n_spikes, &a, &x, &out_spike_times, &out_discrete_spike_times, &post_exp_tau,
                           n_neurons, sample_idx, neuron_idx, max_n_post_spike);

        float cumul_a = 0.0;
        float cumul_b = 0.0;
        float weight;
        int next_i;
        float next_spike;

        for (int i = 0; i < max_n_pre_spike; i++) {
            if (discrete_spike_times[i] == INFINITY) // No spike anymore --> stop
                break;
            weight = spike_weights[i];

            cumul_a += weight * exp_tau_s[i];
            cumul_b += weight * exp_tau[i];


            next_i = i + 1;

            //printf("%f %f\n", spike_times[next_i], discrete_spike_times[next_i]);

            if (next_i < max_n_pre_spike)
                next_spike = discrete_spike_times[next_i]; // we use discrete spike times
            else                                  // replacing this with discrete breaks everything
                next_spike = INFINITY;
            if (compute_spikes(c, n_spikes, a, x, out_spike_times, out_discrete_spike_times, post_exp_tau,
                               cumul_a, &cumul_b, spike_times[i], discrete_spike_times[i], next_spike, delta_theta_tau, tau, time_delta,
                               max_simulation, neuron_idx, max_n_post_spike, sample_idx))
                break; // Buffer full
            //if(out_spike_times[n_spikes[i]] != INFINITY)
            //printf("%d\n", n_spikes[i]);
            //  printf("%f %f\n", out_spike_times[n_spikes[i]], out_discrete_spike_times[n_spikes[i]]);
            //n_spikes[i] is the spike
        }
    }
}
